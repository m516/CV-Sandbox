#include "hip/hip_runtime.h"

#include <iostream>
#include <string>
#include <sstream>
#include <glad.h>
#include <cuda_gl_interop.h>
#include <GLFW/glfw3.h>

#include "cuda_check_error.h"
#include "timer.h"


#define THREADS_PER_BLOCK 128

int windowWidth = 800, windowHeight = 480;

using namespace std;

//A custom, lightweight CUDA implementation of a 2D vector with floats.
class CUDAVec2{
public:
  __device__ CUDAVec2(float x = 0, float y = 0){this->x = x; this->y=y;}
  __device__ CUDAVec2(CUDAVec2& v){x = v.x; y=v.y;}
  float x, y;
  //Get the magnitude of this vector.
  __device__ float mag(){return hypotf(x,y);}
  //Subtract an amount from this vector.
  __device__ void subtract(CUDAVec2 amount){x-=amount.x;y-=amount.y;}
  //Subtract an amount from this vector.
  __device__ void subtract(float x, float y){this->x-=x;this->y-=y;}
};

__global__ void update_surface(hipSurfaceObject_t surface, int textureWidth, int textureHeight, int i)
{
  int xPx = threadIdx.x + blockIdx.x * blockDim.x;
  int yPx = threadIdx.y + blockIdx.y * blockDim.y;

  if(xPx >= textureWidth)
    return;
  if(yPx >= textureHeight)
    return;

  CUDAVec2 normalizedCoordinates((float)xPx / textureWidth, (float)yPx / textureHeight);
  i%=400;
  float theta = (float)i/63.661977236758134307553505349006f;

  bool inCircle = false;

  //Create and find the intersection of a bunch of circles
  int numCircles = 10;
  for(int i = 0; i < numCircles; i++){
    theta+=(float)6.283185307179586476925286766559f/numCircles;
    CUDAVec2 circlePosition(0.5+0.4*cos(theta), 0.5+0.4*sin(theta));
    CUDAVec2 v (normalizedCoordinates);
    v.subtract(circlePosition);
    float r = v.mag();
    if(r<0.1){
      inCircle = true;
      break;
    }
  }


  float red = normalizedCoordinates.x;
  float green = 1.f - normalizedCoordinates.y;
  float blue = 1.f-normalizedCoordinates.x;
  float alpha = 1.f;

  if(inCircle){
    red = 1.f - red;
    green = 1.f - green;
    blue = 1.f - blue;
  }


  uchar4 pixel = { (uint8_t)(red*255),
    (uint8_t)(green*255),
    (uint8_t)(blue*255),
    (uint8_t)(alpha*255)};

  surf2Dwrite(pixel, surface, xPx * sizeof(uchar4), yPx);
}

static void init_opengl(int w, int h) {
  glViewport(0, 0, w, h); // use a screen size of windowWidth x windowHeight

  glMatrixMode(GL_PROJECTION);     // Make a simple 2D projection on the entire window
  glLoadIdentity();
  glOrtho(0.0, w, h, 0.0, 0.0, 100.0);

  glMatrixMode(GL_MODELVIEW);    // Set the matrix mode to object modeling

  glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
  glClearDepth(0.0f);
  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT); // Clear the window
}

/**
 * A helper function for terminating the program
 */
void terminate(int errorCode) {
    cout << "Closing application";
    //Close GLFW
    glfwTerminate();
    //Exit
    exit(errorCode);
}


static void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods) {
  if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS) {
      glfwSetWindowShouldClose(window, GLFW_TRUE);
  }
}

static void resize_callback(GLFWwindow* window, int new_width, int new_height) {
  glViewport(0, 0, windowWidth = new_width, windowHeight = new_height);
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  glOrtho(0.0, windowWidth, windowHeight, 0.0, 0.0, 100.0);
  glMatrixMode(GL_MODELVIEW);
}

static void error_callback(int error, const char* description) {
  fprintf(stderr, "Error: %s\n", description);
}

int main(int argc, char **argv)
{
    // Initialize GLFW, and GLAD, in exactly the same way as project 4.
    GLFWwindow* window;
    glfwSetErrorCallback(error_callback);
    if (!glfwInit()) {
        exit(EXIT_FAILURE);
    }
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 2);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);
    window = glfwCreateWindow(windowWidth, windowHeight, "Simple CUDA + GLFW example", NULL, NULL);
    if (!window) {
        glfwTerminate();
        exit(EXIT_FAILURE);
    }
    glfwSetKeyCallback(window, key_callback);
    glfwSetWindowSizeCallback(window, resize_callback);
    glfwMakeContextCurrent(window);
    glfwSwapInterval(1);
    if (!gladLoadGL()) {
        cerr << "GLAD failed to initialize :(";
        terminate(1);
    }
    init_opengl(windowWidth, windowHeight);
    // End GLAD and GLFW setup

    // Create the OpenGL texture that will be displayed with GLAD and GLFW
    GLuint textureID;
    glGenTextures(1, &textureID);   
    // Bind to our texture handle
    glBindTexture(GL_TEXTURE_2D, textureID);
    // Set texture interpolation methods for minification and magnification
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    // Set texture clamping method
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
    // Create the texture and its attributes
    int textureWidth = windowWidth, textureHeight = windowHeight;
    unsigned char* data = new unsigned char[textureWidth*textureHeight*4]; 
    glTexImage2D(GL_TEXTURE_2D,     // Type of texture
        0,                 // Pyramid level (for mip-mapping) - 0 is the top level
        GL_RGBA,            // Internal colour format to convert to
        windowWidth,          // Image width  i.e. 640 for Kinect in standard mode
        windowHeight,          // Image height i.e. 480 for Kinect in standard mode
        0,                 // Border width in pixels (can either be 1 or 0)
        GL_RGBA, // Input image format (i.e. GL_RGB, GL_RGBA, GL_BGR etc.)
        GL_UNSIGNED_BYTE,  // Image data type
        data);        // The actual image data itself

    //Create the CUDA array and texture reference
    hipArray *bitmap_d;
    hipGraphicsResource *cudaTextureID;
    //Register the GL texture with the CUDA graphics library. A new hipGraphicsResource is created, and its address is placed in cudaTextureID.
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__OPENGL.html#group__CUDART__OPENGL_1g80d12187ae7590807c7676697d9fe03d
    hipGraphicsGLRegisterImage(&cudaTextureID, textureID, GL_TEXTURE_2D,
                                hipGraphicsRegisterFlagsNone);
    cudaCheckError();
    //Map graphics resources for access by CUDA.
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__INTEROP.html#group__CUDART__INTEROP_1gad8fbe74d02adefb8e7efb4971ee6322
    hipGraphicsMapResources(1, &cudaTextureID, 0);
    cudaCheckError();
    //Get the location of the array of pixels that was mapped by the previous function and place that address in bitmap_d
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__INTEROP.html#group__CUDART__INTEROP_1g0dd6b5f024dfdcff5c28a08ef9958031
    hipGraphicsSubResourceGetMappedArray(&bitmap_d, cudaTextureID, 0, 0);
    cudaCheckError();
    //Create a CUDA resource descriptor. This is used to get and set attributes of CUDA resources.
    //This one will tell CUDA how we want the bitmap_surface to be configured.
    //Documentation for the struct: https://docs.nvidia.com/cuda/cuda-runtime-api/structcudaResourceDesc.html#structcudaResourceDesc
    struct hipResourceDesc resDesc;
    //Clear it with 0s so that some flags aren't arbitrarily left at 1s
    memset(&resDesc, 0, sizeof(resDesc));
    //Set the resource type to be an array for convenient processing in the CUDA kernel.
    //List of resTypes: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#group__CUDART__TYPES_1g067b774c0e639817a00a972c8e2c203c
    resDesc.resType = hipResourceTypeArray;
    //Bind the new descriptor with the bitmap created earlier.
    resDesc.res.array.array = bitmap_d;
    //Create a new CUDA surface ID reference.
    //This is really just an unsigned long long.
    //Docuentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#group__CUDART__TYPES_1gbe57cf2ccbe7f9d696f18808dd634c0a
    hipSurfaceObject_t bitmap_surface = 0;
    //Create the surface with the given description. That surface ID is placed in bitmap_surface.
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__SURFACE__OBJECT.html#group__CUDART__SURFACE__OBJECT_1g958899474ab2c5f40d233b524d6c5a01
    hipCreateSurfaceObject(&bitmap_surface, &resDesc);
    cudaCheckError();
  
    dim3 blocks((unsigned int)ceil((float)textureWidth / THREADS_PER_BLOCK), textureHeight);

    //Frame counter
    int i = 0;

    while (!glfwWindowShouldClose(window)) {
      //Start the timer
      Timer stopwatch;
      //Update the texture with the CUDA kernel
      update_surface<<<blocks, THREADS_PER_BLOCK>>>(bitmap_surface, textureWidth, textureHeight, i++);
      //Print elapsed time occasionally
      if(i%10==0){
        float fps = 1.f/stopwatch.getElapsedSeconds();
        ostringstream myString;
        myString << "Simple CUDA + GLFW example (";
        myString.precision(2);
        myString << std::fixed << fps;
        myString << " FPS)";
        glfwSetWindowTitle(window, myString.str().c_str());
      }
      cudaCheckError();
      hipDeviceSynchronize();
      cudaCheckError();

      //Render the results on the screen buffer
      //Clear color and depth buffers
      glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
      //Begin drawing
      glMatrixMode(GL_MODELVIEW);     //Operate on model-view matrix
      glEnable(GL_TEXTURE_2D);
      //Bind to our texture handle
      glBindTexture(GL_TEXTURE_2D, textureID);
      //Draw a quad
      glBegin(GL_QUADS);
      glTexCoord2i(0, 0); glVertex2i(0, 0);
      glTexCoord2i(0, 1); glVertex2i(0, windowHeight);
      glTexCoord2i(1, 1); glVertex2i(windowWidth, windowHeight);
      glTexCoord2i(1, 0); glVertex2i(windowWidth, 0);
      glEnd();
      glDisable(GL_TEXTURE_2D);
      
      //Blit rendered contents on the screen and poll events.
      glfwSwapBuffers(window);
      glfwPollEvents();
    }

    glfwDestroyWindow(window);
    glfwTerminate();

    exit(EXIT_SUCCESS);
  


  return 0;
}
