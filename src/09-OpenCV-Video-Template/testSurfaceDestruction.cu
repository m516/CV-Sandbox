

#include <glad/glad.h>
#include <GLFW/glfw3.h>

#include <cudaGL.h>
#include <cuda_gl_interop.h>

#include <iostream>



#define cudaCheckError() { \
    hipError_t err = hipGetLastError(); \
    if(err != hipSuccess) { \
      printf("Cuda error: %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
      exit(1); \
    } \
  }

void createTextureSurfacePair(int width, int height, uint8_t* const data, GLuint& textureOut, hipGraphicsResource_t& graphicsResourceOut, hipSurfaceObject_t& surfaceOut) {

    // Create the OpenGL texture that will be displayed with GLAD and GLFW
    glGenTextures(1, &textureOut);
    // Bind to our texture handle
    glBindTexture(GL_TEXTURE_2D, textureOut);
    // Set texture interpolation methods for minification and magnification
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    // Set texture clamping method
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
    // Create the texture and its attributes
    glTexImage2D(GL_TEXTURE_2D,     // Type of texture
        0,                // Pyramid level (for mip-mapping) - 0 is the top level
        GL_RGBA,          // Internal color format to convert to
        width,            // Image width  i.e. 640 for Kinect in standard mode
        height,           // Image height i.e. 480 for Kinect in standard mode
        0,                // Border width in pixels (can either be 1 or 0)
        GL_BGR,          // Input image format (i.e. GL_RGB, GL_RGBA, GL_BGR etc.)
        GL_UNSIGNED_BYTE, // Image data type.
        data);            // The actual image data itself
    //Note that the type of this texture is an RGBA UNSIGNED_BYTE type. When CUDA surfaces
    //are synchronized with OpenGL textures, the surfaces will be of the same type.
    //They won't know or care about their data types though, for they are all just byte arrays
    //at heart. So be careful to ensure that any CUDA kernel that handles a CUDA surface
    //uses it as an appropriate type. You will see that the update_surface kernel (defined 
    //above) treats each pixel as four unsigned bytes along the X-axis: one for red, green, blue,
    //and alpha respectively.

    //Create the CUDA array and texture reference
    hipArray* bitmap_d;
    //Register the GL texture with the CUDA graphics library. A new hipGraphicsResource is created, and its address is placed in cudaTextureID.
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__OPENGL.html#group__CUDART__OPENGL_1g80d12187ae7590807c7676697d9fe03d
    hipGraphicsGLRegisterImage(&graphicsResourceOut, textureOut, GL_TEXTURE_2D,
        hipGraphicsRegisterFlagsNone);
    cudaCheckError();
    //Map graphics resources for access by CUDA.
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__INTEROP.html#group__CUDART__INTEROP_1gad8fbe74d02adefb8e7efb4971ee6322
    hipGraphicsMapResources(1, &graphicsResourceOut, 0);
    cudaCheckError();
    //Get the location of the array of pixels that was mapped by the previous function and place that address in bitmap_d
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__INTEROP.html#group__CUDART__INTEROP_1g0dd6b5f024dfdcff5c28a08ef9958031
    hipGraphicsSubResourceGetMappedArray(&bitmap_d, graphicsResourceOut, 0, 0);
    cudaCheckError();
    //Create a CUDA resource descriptor. This is used to get and set attributes of CUDA resources.
    //This one will tell CUDA how we want the bitmap_surface to be configured.
    //Documentation for the struct: https://docs.nvidia.com/cuda/cuda-runtime-api/structcudaResourceDesc.html#structcudaResourceDesc
    struct hipResourceDesc resDesc;
    //Clear it with 0s so that some flags aren't arbitrarily left at 1s
    memset(&resDesc, 0, sizeof(resDesc));
    //Set the resource type to be an array for convenient processing in the CUDA kernel.
    //List of resTypes: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#group__CUDART__TYPES_1g067b774c0e639817a00a972c8e2c203c
    resDesc.resType = hipResourceTypeArray;
    //Bind the new descriptor with the bitmap created earlier.
    resDesc.res.array.array = bitmap_d;
    //Create a new CUDA surface ID reference.
    //This is really just an unsigned long long.
    //Docuentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#group__CUDART__TYPES_1gbe57cf2ccbe7f9d696f18808dd634c0a
    surfaceOut = 0;
    //Create the surface with the given description. That surface ID is placed in bitmap_surface.
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__SURFACE__OBJECT.html#group__CUDART__SURFACE__OBJECT_1g958899474ab2c5f40d233b524d6c5a01
    hipCreateSurfaceObject(&surfaceOut, &resDesc);
    cudaCheckError();
}


void initGL() {

    // Setup window
    if (!glfwInit())
        return;

    // Decide GL+GLSL versions
#if __APPLE__
    // GL 3.2 + GLSL 150
    const char* glsl_version = "#version 150";
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 2);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);  // 3.2+ only
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);            // Required on Mac
#else
    // GL 3.0 + GLSL 130
    const char* glsl_version = "#version 130";
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);
    //glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);  // 3.2+ only
    //glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);            // 3.0+ only
#endif

    // Create window with graphics context
    GLFWwindow* currentGLFWWindow = glfwCreateWindow(1280, 720, "GLFW Window", NULL, NULL);
    if (currentGLFWWindow == NULL)
        return;
    glfwMakeContextCurrent(currentGLFWWindow);
    glfwSwapInterval(3); // Enable vsync

    if (!gladLoadGL()) {
        // GLAD failed
        printf( "GLAD failed to initialize :(" );
        return;
    }
}


int main() {
    initGL();

    int size = 500;

    uint8_t* data = new uint8_t[size * size * 3]; //dummy 100x100 RGB image

    hipSurfaceObject_t a;
    hipGraphicsResource_t b;
    GLuint c;

    for (int i = 0; i < 10000; i++) {
/*------ATTEMPT TO CREATE CUDA SURFACE AND OPENGL TEXTURE------------*/
        createTextureSurfacePair(size, size, data, c, b, a);

/*------ATTEMPT TO DESTROY CUDA SURFACE AND OPENGL TEXTURE------------*/
//https://stackoverflow.com/questions/63455251/how-to-destroy-cuda-graphics-datatypes
        //Destroy surface
        hipDestroySurfaceObject(a);
        //Destroy graphics resource
        hipGraphicsUnmapResources(1, &b);
        hipGraphicsUnregisterResource(b);
        //Destroy texture
        glDeleteTextures(1, &c);

        if (i % 100 == 0) printf("Iteration %d\n", i);
    }
}