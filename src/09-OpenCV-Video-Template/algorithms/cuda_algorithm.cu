#include "cuda_algorithm.cuh"
#include <imgui/imgui.h>
#include <iostream>
#include <string>



#define cudaCheckError() { \
    hipError_t err = hipGetLastError(); \
    if(err != hipSuccess) { \
      printf("Cuda error: %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
      exit(1); \
    } \
  }


void CUDAVisionAlgorithm::createTextureSurfacePair(const CUDAVisionAlgorithm::ImgDim& dimensions, uint8_t* const data, GLuint& textureOut, hipGraphicsResource_t& graphicsResourceOut, hipSurfaceObject_t& surfaceOut){

    // Create the OpenGL texture that will be displayed with GLAD and GLFW
    glGenTextures(1, &textureOut);   
    // Bind to our texture handle
    glBindTexture(GL_TEXTURE_2D, textureOut);
    // Set texture interpolation methods for minification and magnification
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    // Set texture clamping method
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
    // Create the texture and its attributes
    glTexImage2D(GL_TEXTURE_2D,     // Type of texture
        0,                // Pyramid level (for mip-mapping) - 0 is the top level
        GL_RGBA,          // Internal color format to convert to
        dimensions.width,            // Image width  i.e. 640 for Kinect in standard mode
        dimensions.height,           // Image height i.e. 480 for Kinect in standard mode
        0,                // Border width in pixels (can either be 1 or 0)
        GL_BGR,          // Input image format (i.e. GL_RGB, GL_RGBA, GL_BGR etc.)
        GL_UNSIGNED_BYTE, // Image data type.
        data);            // The actual image data itself
    //Note that the type of this texture is an RGBA UNSIGNED_BYTE type. When CUDA surfaces
    //are synchronized with OpenGL textures, the surfaces will be of the same type.
    //They won't know or care about their data types though, for they are all just byte arrays
    //at heart. So be careful to ensure that any CUDA kernel that handles a CUDA surface
    //uses it as an appropriate type. You will see that the update_surface kernel (defined 
    //above) treats each pixel as four unsigned bytes along the X-axis: one for red, green, blue,
    //and alpha respectively.

    //Create the CUDA array and texture reference
    hipArray *bitmap_d;
    //Register the GL texture with the CUDA graphics library. A new hipGraphicsResource is created, and its address is placed in cudaTextureID.
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__OPENGL.html#group__CUDART__OPENGL_1g80d12187ae7590807c7676697d9fe03d
    hipGraphicsGLRegisterImage(&graphicsResourceOut, textureOut, GL_TEXTURE_2D,
                                hipGraphicsRegisterFlagsNone);
    cudaCheckError();
    //Map graphics resources for access by CUDA.
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__INTEROP.html#group__CUDART__INTEROP_1gad8fbe74d02adefb8e7efb4971ee6322
    hipGraphicsMapResources(1, &graphicsResourceOut, 0);
    cudaCheckError();
    //Get the location of the array of pixels that was mapped by the previous function and place that address in bitmap_d
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__INTEROP.html#group__CUDART__INTEROP_1g0dd6b5f024dfdcff5c28a08ef9958031
    hipGraphicsSubResourceGetMappedArray(&bitmap_d, graphicsResourceOut, 0, 0);
    cudaCheckError();
    //Create a CUDA resource descriptor. This is used to get and set attributes of CUDA resources.
    //This one will tell CUDA how we want the bitmap_surface to be configured.
    //Documentation for the struct: https://docs.nvidia.com/cuda/cuda-runtime-api/structcudaResourceDesc.html#structcudaResourceDesc
    struct hipResourceDesc resDesc;
    //Clear it with 0s so that some flags aren't arbitrarily left at 1s
    memset(&resDesc, 0, sizeof(resDesc));
    //Set the resource type to be an array for convenient processing in the CUDA kernel.
    //List of resTypes: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#group__CUDART__TYPES_1g067b774c0e639817a00a972c8e2c203c
    resDesc.resType = hipResourceTypeArray;
    //Bind the new descriptor with the bitmap created earlier.
    resDesc.res.array.array = bitmap_d;
    //Create a new CUDA surface ID reference.
    //This is really just an unsigned long long.
    //Docuentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#group__CUDART__TYPES_1gbe57cf2ccbe7f9d696f18808dd634c0a
    surfaceOut = 0;
    //Create the surface with the given description. That surface ID is placed in bitmap_surface.
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__SURFACE__OBJECT.html#group__CUDART__SURFACE__OBJECT_1g958899474ab2c5f40d233b524d6c5a01
    hipCreateSurfaceObject(&surfaceOut, &resDesc);
    cudaCheckError();
}

void CUDAVisionAlgorithm::setInput(const Mat& input)
{

    //Empty Mats cause problems in textures and surfaces.
    if (input.empty()) return;


    //Same-size images don't need texture regeneration, so skip that.
    if(input.rows == imageInputDimensions.height && input.cols == imageInputDimensions.width){

        //Use the input surface's CUDAResourceDesc to gain access to the surface data array
        struct hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        cudaGetSurfaceObjectResourceDesc(&resDesc, d_imageInputTexture);
        cudaCheckError();

        //Copy the data from the input array to the surface
        hipMemcpyToArray(resDesc.res.array.array, 0, 0, input.data, imageInputDimensions.width* imageInputDimensions.height*3, hipMemcpyHostToDevice);
        cudaCheckError();

        //Set status flags
        surfacesInitialized = true;
        alreadyProcessed = false;

        return;
    }


    //Clear everything that originally existed in the texture/surface
    destroyEverything();

    //Expect a BGR matrix
    if(input.type()!=CV_8UC3){
        std::string r;

        uchar depth = input.type() & CV_MAT_DEPTH_MASK;
        uchar chans = 1 + (input.type() >> CV_CN_SHIFT);

        switch (depth) {
        case CV_8U:  r = "8U"; break;
        case CV_8S:  r = "8S"; break;
        case CV_16U: r = "16U"; break;
        case CV_16S: r = "16S"; break;
        case CV_32S: r = "32S"; break;
        case CV_32F: r = "32F"; break;
        case CV_64F: r = "64F"; break;
        default:     r = "User"; break;
        }

        r += "C";
        r += (chans + '0');

        std::cerr << "Unexpected Mat Type: " << r << std::endl;
        throw std::invalid_argument("Invalid OpenCV matrix type. Expecting a BGR matrix (3-channel)");
    }

    //Get the size of the Mat and place it here.
    imageInputDimensions.width=input.cols;
    imageInputDimensions.height=input.rows;

    //Set the dimensions of the output matrix.
    setOutputDimensions();

    //Create the input surface/texture pair
    createTextureSurfacePair(imageInputDimensions, input.data, imageInputTexture, d_imageInputGraphicsResource, d_imageInputTexture);

    //Create the output surface/texture pair
    uint8_t* outData = new uint8_t[imageOutputDimensions.width * imageOutputDimensions.height * 3];
    createTextureSurfacePair(imageOutputDimensions, outData, imageOutputTexture, d_imageOutputGraphicsResource, d_imageOutputTexture);
    delete outData;

    //Set status flags
    surfacesInitialized = true;
    alreadyProcessed = false;
}

void CUDAVisionAlgorithm::destroyEverything(){
    if (surfacesInitialized) {

        //Input image CUDA surface
        hipDestroySurfaceObject(d_imageInputTexture);
        hipGraphicsUnmapResources(1, &d_imageInputGraphicsResource);
        hipGraphicsUnregisterResource(d_imageInputGraphicsResource);
        d_imageInputTexture = 0;

        //Output image CUDA surface
        hipDestroySurfaceObject(d_imageOutputTexture);
        hipGraphicsUnmapResources(1, &d_imageOutputGraphicsResource);
        hipGraphicsUnregisterResource(d_imageOutputGraphicsResource);
        d_imageOutputTexture = 0;

        //Input image GL texture
        glDeleteTextures(1, &imageInputTexture);
        imageInputTexture = 0;

        //Output image GL texture
        glDeleteTextures(1, &imageOutputTexture);
        imageOutputTexture = 0;

        surfacesInitialized = false;
    }
}

void CUDAVisionAlgorithm::getOutput(Mat& output)
{
    //TODO
    //output = cv::Mat(2, 4, CV_32F, dummy_query_data);
}

void CUDAVisionAlgorithm::addToGUI(){

    //Add a process button
    if (ImGui::Button("Process me!")) process();
    ImGui::Separator();

    //Synchronize the CUDA surfaces with the OpenGL textures, so the textures have the same data their surfaces.
    if(dirty){
        hipDeviceSynchronize();
        cudaCheckError();
        dirty = false;
    }

    if (imageInputTexture!=0) {
        ImGui::Image((ImTextureID)imageInputTexture, ImVec2(imageInputDimensions.width, imageInputDimensions.height));
    }
    else {
        ImGui::Text("No input data to show here.");
    }

    if (imageOutputTexture != 0) {
        ImGui::Image((ImTextureID)imageOutputTexture, ImVec2(imageOutputDimensions.width, imageOutputDimensions.height));
    }
    else {
        ImGui::Text("No output data to show here.");
    }
}