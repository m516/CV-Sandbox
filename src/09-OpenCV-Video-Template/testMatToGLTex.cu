#include "hip/hip_runtime.h"


#include <glad/glad.h>
#include <GLFW/glfw3.h>

#include <cudaGL.h>
#include <cuda_gl_interop.h>

#include <iostream>


/** Macro for checking if CUDA has problems */
#define cudaCheckError() { \
    hipError_t err = hipGetLastError(); \
    if(err != hipSuccess) { \
      printf("Cuda error: %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
      exit(1); \
    } \
  }


/*Window dimensions*/
const int windowWidth = 1280, windowHeight = 720;
/*Window address*/
GLFWwindow* currentGLFWWindow = 0;


/**
 * A simple image processing kernel that copies the inverted data from the input surface to the output surface.
 */
__global__ void kernel(hipSurfaceObject_t input, hipSurfaceObject_t output, int width, int height) {

    //Get the pixel index
    unsigned int xPx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int yPx = threadIdx.y + blockIdx.y * blockDim.y;


    //Don't do any computation if this thread is outside of the surface bounds.
    if (xPx >= width || yPx >= height) return;

    //Copy the contents of input to output.
    uchar4 pixel = { 255,128,0,255 };
    //Read a pixel from the input. Disable to default to the flat orange color above
    surf2Dread<uchar4>(&pixel, input, xPx * sizeof(uchar4), yPx, hipBoundaryModeClamp);

    //Invert the color
    pixel.x = ~pixel.x;
    pixel.y = ~pixel.y;
    pixel.z = ~pixel.z;

    //Write the new pixel color to the 
    surf2Dwrite(pixel, output, xPx * sizeof(uchar4), yPx);
}

class Processor {
public:
    void setInput( uint8_t* const data, int imageWidth, int imageHeight);
    void processData();
    GLuint getInputTexture();
    GLuint getOutputTexture();
    void writeOutputTo(uint8_t* destination);
private:
    /**
    * @brief True if the textures and surfaces are initialized.
    *
    * Prevents memory leaks
    */
    bool surfacesInitialized = false;
    /**
     * @brief The width and height of a texture/surface pair.
     *
     */
    struct ImgDim { int width, height; };
    /**
     * @brief Creates a CUDA surface object, CUDA resource, and OpenGL texture from some data.
     */
    void createTextureSurfacePair(const ImgDim& dimensions, uint8_t* const data, GLuint& textureOut, hipGraphicsResource_t& graphicsResourceOut, hipSurfaceObject_t& surfaceOut);
    /**
     * @brief Destroys every CUDA surface object, CUDA resource, and OpenGL texture created by this instance.
     */
    void destroyEverything();
    /**
     * @brief The dimensions of an image and its corresponding texture.
     *
     */
    ImgDim imageInputDimensions, imageOutputDimensions;
    /**
     * @brief A CUDA surface that can be read to, written from, or synchronized with a Mat or
     * OpenGL texture
     *
     */
    hipSurfaceObject_t d_imageInputTexture = 0, d_imageOutputTexture = 0;
    /**
     * @brief A CUDA resource that's bound to an array in CUDA memory
     */
    hipGraphicsResource_t d_imageInputGraphicsResource, d_imageOutputGraphicsResource;
    /**
     * @brief A renderable OpenGL texture that is synchronized with the CUDA data
     * @see d_imageInputTexture, d_imageOutputTexture
     */
    GLuint imageInputTexture = 0, imageOutputTexture = 0;
    /** Returns true if nothing can be rendered */
    bool empty() { return imageInputTexture == 0; }

};


void Processor::setInput(uint8_t* const data, int imageWidth, int imageHeight)
{


    //Same-size images don't need texture regeneration, so skip that.
    if (imageHeight == imageInputDimensions.height && imageWidth == imageInputDimensions.width) {


        /*
        Possible shortcut: we know the input is the same size as the texture and CUDA surface object.
        So instead of destroying the surface and texture, why not just overwrite them?

        That's what I try to do in the following block, but because "data" is BGR and the texture
        is RGBA, the channels get all messed up.
        */

        /*
        //Use the input surface's CUDAResourceDesc to gain access to the surface data array
        struct hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        cudaGetSurfaceObjectResourceDesc(&resDesc, d_imageInputTexture);
        cudaCheckError();

        //Copy the data from the input array to the surface
        hipMemcpyToArray(resDesc.res.array.array, 0, 0, input.data, imageInputDimensions.width * imageInputDimensions.height * 3, hipMemcpyHostToDevice);
        cudaCheckError();

        //Set status flags
        surfacesInitialized = true;

        return;
        */
    }


    //Clear everything that originally existed in the texture/surface
    destroyEverything();

    //Get the size of the image and place it here.
    imageInputDimensions.width = imageWidth;
    imageInputDimensions.height = imageHeight;
    imageOutputDimensions.width = imageWidth;
    imageOutputDimensions.height = imageHeight;

    //Create the input surface/texture pair
    createTextureSurfacePair(imageInputDimensions, data, imageInputTexture, d_imageInputGraphicsResource, d_imageInputTexture);

    //Create the output surface/texture pair
    uint8_t* outData = new uint8_t[imageOutputDimensions.width * imageOutputDimensions.height * 3];
    createTextureSurfacePair(imageOutputDimensions, outData, imageOutputTexture, d_imageOutputGraphicsResource, d_imageOutputTexture);
    delete outData;

    //Set status flags
    surfacesInitialized = true;
}

void Processor::processData()
{
    const int threadsPerBlock = 128;

    //Call the algorithm

    //Set the number of blocks to call the kernel with.
    dim3 blocks((unsigned int)ceil((float)imageInputDimensions.width / threadsPerBlock), imageInputDimensions.height);
    kernel <<<blocks, threadsPerBlock >>> (d_imageInputTexture, d_imageOutputTexture, imageInputDimensions.width, imageInputDimensions.height);

    //Sync the surface with the texture
    hipDeviceSynchronize();
    cudaCheckError();
}

GLuint Processor::getInputTexture()
{
    return imageInputTexture;
}

GLuint Processor::getOutputTexture()
{
    return imageOutputTexture;
}

void Processor::writeOutputTo(uint8_t* destination)
{
    //Haven't figured this out yet
}

void Processor::createTextureSurfacePair(const Processor::ImgDim& dimensions, uint8_t* const data, GLuint& textureOut, hipGraphicsResource_t& graphicsResourceOut, hipSurfaceObject_t& surfaceOut) {

    // Create the OpenGL texture that will be displayed with GLAD and GLFW
    glGenTextures(1, &textureOut);
    // Bind to our texture handle
    glBindTexture(GL_TEXTURE_2D, textureOut);
    // Set texture interpolation methods for minification and magnification
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    // Set texture clamping method
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
    // Create the texture and its attributes
    glTexImage2D(GL_TEXTURE_2D,     // Type of texture
        0,                // Pyramid level (for mip-mapping) - 0 is the top level
        GL_RGBA,          // Internal color format to convert to
        dimensions.width,            // Image width  i.e. 640 for Kinect in standard mode
        dimensions.height,           // Image height i.e. 480 for Kinect in standard mode
        0,                // Border width in pixels (can either be 1 or 0)
        GL_BGR,          // Input image format (i.e. GL_RGB, GL_RGBA, GL_BGR etc.)
        GL_UNSIGNED_BYTE, // Image data type.
        data);            // The actual image data itself
    //Note that the type of this texture is an RGBA UNSIGNED_BYTE type. When CUDA surfaces
    //are synchronized with OpenGL textures, the surfaces will be of the same type.
    //They won't know or care about their data types though, for they are all just byte arrays
    //at heart. So be careful to ensure that any CUDA kernel that handles a CUDA surface
    //uses it as an appropriate type. You will see that the update_surface kernel (defined 
    //above) treats each pixel as four unsigned bytes along the X-axis: one for red, green, blue,
    //and alpha respectively.

    //Create the CUDA array and texture reference
    hipArray* bitmap_d;
    //Register the GL texture with the CUDA graphics library. A new hipGraphicsResource is created, and its address is placed in cudaTextureID.
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__OPENGL.html#group__CUDART__OPENGL_1g80d12187ae7590807c7676697d9fe03d
    hipGraphicsGLRegisterImage(&graphicsResourceOut, textureOut, GL_TEXTURE_2D,
        hipGraphicsRegisterFlagsNone);
    cudaCheckError();
    //Map graphics resources for access by CUDA.
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__INTEROP.html#group__CUDART__INTEROP_1gad8fbe74d02adefb8e7efb4971ee6322
    hipGraphicsMapResources(1, &graphicsResourceOut, 0);
    cudaCheckError();
    //Get the location of the array of pixels that was mapped by the previous function and place that address in bitmap_d
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__INTEROP.html#group__CUDART__INTEROP_1g0dd6b5f024dfdcff5c28a08ef9958031
    hipGraphicsSubResourceGetMappedArray(&bitmap_d, graphicsResourceOut, 0, 0);
    cudaCheckError();
    //Create a CUDA resource descriptor. This is used to get and set attributes of CUDA resources.
    //This one will tell CUDA how we want the bitmap_surface to be configured.
    //Documentation for the struct: https://docs.nvidia.com/cuda/cuda-runtime-api/structcudaResourceDesc.html#structcudaResourceDesc
    struct hipResourceDesc resDesc;
    //Clear it with 0s so that some flags aren't arbitrarily left at 1s
    memset(&resDesc, 0, sizeof(resDesc));
    //Set the resource type to be an array for convenient processing in the CUDA kernel.
    //List of resTypes: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#group__CUDART__TYPES_1g067b774c0e639817a00a972c8e2c203c
    resDesc.resType = hipResourceTypeArray;
    //Bind the new descriptor with the bitmap created earlier.
    resDesc.res.array.array = bitmap_d;
    //Create a new CUDA surface ID reference.
    //This is really just an unsigned long long.
    //Docuentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#group__CUDART__TYPES_1gbe57cf2ccbe7f9d696f18808dd634c0a
    surfaceOut = 0;
    //Create the surface with the given description. That surface ID is placed in bitmap_surface.
    //Documentation: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__SURFACE__OBJECT.html#group__CUDART__SURFACE__OBJECT_1g958899474ab2c5f40d233b524d6c5a01
    hipCreateSurfaceObject(&surfaceOut, &resDesc);
    cudaCheckError();
}

void Processor::destroyEverything()
{
    if (surfacesInitialized) {

        //Input image CUDA surface
        hipDestroySurfaceObject(d_imageInputTexture);
        hipGraphicsUnmapResources(1, &d_imageInputGraphicsResource);
        hipGraphicsUnregisterResource(d_imageInputGraphicsResource);
        d_imageInputTexture = 0;

        //Output image CUDA surface
        hipDestroySurfaceObject(d_imageOutputTexture);
        hipGraphicsUnmapResources(1, &d_imageOutputGraphicsResource);
        hipGraphicsUnregisterResource(d_imageOutputGraphicsResource);
        d_imageOutputTexture = 0;

        //Input image GL texture
        glDeleteTextures(1, &imageInputTexture);
        imageInputTexture = 0;

        //Output image GL texture
        glDeleteTextures(1, &imageOutputTexture);
        imageOutputTexture = 0;

        surfacesInitialized = false;
    }
}


/** A way to initialize OpenGL with GLFW and GLAD */
void initGL() {

    // Setup window
    if (!glfwInit())
        return;

    // Decide GL+GLSL versions
#if __APPLE__
    // GL 3.2 + GLSL 150
    const char* glsl_version = "#version 150";
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 2);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);  // 3.2+ only
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);            // Required on Mac
#else
    // GL 3.0 + GLSL 130
    const char* glsl_version = "#version 130";
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);
    //glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);  // 3.2+ only
    //glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);            // 3.0+ only
#endif

    // Create window with graphics context
    currentGLFWWindow = glfwCreateWindow(windowWidth, windowHeight, "Output image (OpenGL + GLFW)", NULL, NULL);
    if (currentGLFWWindow == NULL)
        return;
    glfwMakeContextCurrent(currentGLFWWindow);
    glfwSwapInterval(3); // Enable vsync

    if (!gladLoadGL()) {
        // GLAD failed
        printf( "GLAD failed to initialize :(" );
        return;
    }

    //Change GL settings
    glViewport(0, 0, windowWidth, windowHeight); // use a screen size of WIDTH x HEIGHT

    glMatrixMode(GL_PROJECTION);     // Make a simple 2D projection on the entire window
    glLoadIdentity();
    glOrtho(0.0, windowWidth, windowHeight, 0.0, 0.0, 100.0);

    glMatrixMode(GL_MODELVIEW);    // Set the matrix mode to object modeling

    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
    glClearDepth(0.0f);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT); // Clear the window
}

/** Renders the textures on the GLFW window and requests GLFW to update */
void showTextures(GLuint top, GLuint bottom) {
    // Clear color and depth buffers
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glMatrixMode(GL_MODELVIEW);     // Operate on model-view matrix

    glBindTexture(GL_TEXTURE_2D, top);
    /* Draw top quad */
    glEnable(GL_TEXTURE_2D);
    glBegin(GL_QUADS);
    glTexCoord2i(0, 0); glVertex2i(0, 0);
    glTexCoord2i(0, 1); glVertex2i(0, windowHeight/2);
    glTexCoord2i(1, 1); glVertex2i(windowWidth, windowHeight / 2);
    glTexCoord2i(1, 0); glVertex2i(windowWidth, 0);
    glEnd();
    glDisable(GL_TEXTURE_2D);
    /* Draw top quad */
    glBindTexture(GL_TEXTURE_2D, bottom);
    glEnable(GL_TEXTURE_2D);
    glBegin(GL_QUADS);
    glTexCoord2i(0, 0); glVertex2i(0, windowHeight / 2);
    glTexCoord2i(0, 1); glVertex2i(0, windowHeight);
    glTexCoord2i(1, 1); glVertex2i(windowWidth, windowHeight);
    glTexCoord2i(1, 0); glVertex2i(windowWidth, windowHeight / 2);
    glEnd();
    glDisable(GL_TEXTURE_2D);


    glfwSwapBuffers(currentGLFWWindow);
    glfwPollEvents();
}


int main() {
    initGL();

    int imageWidth = windowWidth;
    int imageHeight = windowHeight / 2;

    uint8_t* imageData = new uint8_t[imageWidth * imageHeight * 3];

    Processor p;

    while (!glfwWindowShouldClose(currentGLFWWindow))
    {
        //Process the image here
        p.setInput(imageData, imageWidth, imageHeight);
        p.processData();
        showTextures(p.getInputTexture(), p.getOutputTexture());
    }
}

